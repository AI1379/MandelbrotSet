#include "hip/hip_runtime.h"
//
// Created by Renatus Madrigal on 3/3/2025.
//

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include "mandelbrot/MandelbrotSetCuda.h"

#define CHECK_CUDA(err)                                                                                                \
    do {                                                                                                               \
        if (err != hipSuccess) {                                                                                      \
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));                                              \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    } while (0)

namespace Mandelbrot {
    constexpr auto MAX_ITERATIONS = MandelbrotSetCuda::MAX_ITERATIONS;

    __constant__ static uchar3 colors[MAX_ITERATIONS + 1];
    static bool initialized = false;

    void initialize() {
        if (initialized)
            return;
        // TODO: Figure out why this is necessary
        static uchar3 colors_temp[MAX_ITERATIONS + 1];
        initialized = true;
        cv::Mat hsv(1, MAX_ITERATIONS, CV_8UC3);
        for (int n = 0; n < MAX_ITERATIONS; ++n) {
            const double hue = 180 * fmod(n * 0.3, 1.0);
            hsv.at<cv::Vec3b>(0, n) = cv::Vec3b(hue, 255, 255);
        }
        cvtColor(hsv, hsv, cv::COLOR_HSV2BGR);

        for (int n = 0; n < MAX_ITERATIONS; ++n) {
            auto color = hsv.at<cv::Vec3b>(0, n);
            colors_temp[n] = {color[0], color[1], color[2]};
        }
        colors_temp[MAX_ITERATIONS] = {0, 0, 0};

        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(colors), colors_temp, sizeof(colors_temp)));
    }

    __global__ void mandelbrotKernel(uchar3 *image, // NOLINT
                                     size_t width, size_t height, // NOLINT
                                     double x_min, double x_max, // NOLINT
                                     double y_min, double y_max) {
        constexpr auto ESCAPE_RADIUS_SQ = MandelbrotSetCuda::ESCAPE_RADIUS_SQ;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        }
        const double cr = x_min + (x_max - x_min) * x / width;
        const double ci = y_min + (y_max - y_min) * y / height;

        double zr = 0.0, zi = 0.0;
        int n = 0;
        while (n < MAX_ITERATIONS) {
            const double zr2 = zr * zr;
            const double zi2 = zi * zi;
            if (zr2 + zi2 > ESCAPE_RADIUS_SQ)
                break;

            const double zr_temp = zr2 - zi2 + cr;
            zi = 2 * zr * zi + ci;
            zr = zr_temp;
            ++n;
        }
        const int idx = y * width + x;
        image[idx] = colors[n];
    }

    cv::Mat MandelbrotSetCuda::generateImpl() const {
        initialize();

        uchar3 *d_image;
        CHECK_CUDA(hipMalloc(&d_image, width_ * height_ * sizeof(uchar3)));

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width_ + block.x - 1) / block.x, (height_ + block.y - 1) / block.y);

        mandelbrotKernel<<<grid, block>>>(d_image, width_, height_, x_min_, x_max_, y_min_, y_max_);
        CHECK_CUDA(hipGetLastError());

        cv::Mat image(height_, width_, CV_8UC3);
        CHECK_CUDA(hipMemcpy(image.data, d_image, width_ * height_ * sizeof(uchar3), hipMemcpyDeviceToHost));

        CHECK_CUDA(hipFree(d_image));

        return image;
    }


} // namespace Mandelbrot
