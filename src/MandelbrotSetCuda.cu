#include "hip/hip_runtime.h"
//
// Created by Renatus Madrigal on 3/3/2025.
//

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include "ExtendedDouble.cuh"
#include "MandelbrotSetCuda.h"

#define CHECK_CUDA(err)                                                                                                \
    do {                                                                                                               \
        if (err != hipSuccess) {                                                                                      \
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));                                              \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    } while (0)

namespace Mandelbrot {
    constexpr bool USE_EXTENDED_DOUBLE = true;
    using ComputeDouble = std::conditional_t<USE_EXTENDED_DOUBLE, ExtendedDouble, double>;

    constexpr auto MAX_ITERATIONS = MandelbrotSetCuda::MAX_ITERATIONS;

    __constant__ static uchar3 colors[MAX_ITERATIONS + 1];
    static bool initialized = false;

    void initialize() {
        if (initialized)
            return;
        // TODO: Figure out why this is necessary
        static uchar3 colors_temp[MAX_ITERATIONS + 1];
        initialized = true;
        cv::Mat hsv(1, MAX_ITERATIONS, CV_8UC3);
        for (int n = 0; n < MAX_ITERATIONS; ++n) {
            const double hue = 180 * fmod(n * 0.3, 1.0);
            hsv.at<cv::Vec3b>(0, n) = cv::Vec3b(hue, 255, 255);
        }
        cvtColor(hsv, hsv, cv::COLOR_HSV2BGR);

        for (int n = 0; n < MAX_ITERATIONS; ++n) {
            auto color = hsv.at<cv::Vec3b>(0, n);
            colors_temp[n] = {color[0], color[1], color[2]};
        }
        colors_temp[MAX_ITERATIONS] = {0, 0, 0};

        CHECK_CUDA(hipMemcpyToSymbol(HIP_SYMBOL(colors), colors_temp, sizeof(colors_temp)));
    }

    __global__ void mandelbrotKernel(uchar3 *image, // NOLINT
                                     size_t width, size_t height, // NOLINT
                                     double x_min, double x_max, // NOLINT
                                     double y_min, double y_max) {
        constexpr auto ESCAPE_RADIUS_SQ = MandelbrotSetCuda::ESCAPE_RADIUS_SQ;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        }
        const ComputeDouble cr{x_min + (x_max - x_min) * x / width};
        const ComputeDouble ci{y_min + (y_max - y_min) * y / height};

        ComputeDouble zr{0.0}, zi{0.0};
        int n = 0;
        while (n < MAX_ITERATIONS) {
            const ComputeDouble zr2 = zr * zr;
            const ComputeDouble zi2 = zi * zi;
            if (zr2 + zi2 > ESCAPE_RADIUS_SQ)
                break;

            const ComputeDouble zr_temp = zr2 - zi2 + cr;
            zi = zr * zi * 2 + ci;
            zr = zr_temp;
            ++n;
        }
        const int idx = y * width + x;
        image[idx] = colors[n];
    }

    __global__ void mandelbrotKernelWithoutColor(float *image, // NOLINT
                                                 size_t width, size_t height, // NOLINT
                                                 double x_min, double x_max, // NOLINT
                                                 double y_min, double y_max) {
        constexpr auto ESCAPE_RADIUS_SQ = MandelbrotSetCuda::ESCAPE_RADIUS_SQ;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        }
        const ComputeDouble cr{x_min + (x_max - x_min) * x / width};
        const ComputeDouble ci{y_min + (y_max - y_min) * y / height};

        ComputeDouble zr{0.0}, zi{0.0};
        unsigned int n = 0;
        while (n < MAX_ITERATIONS) {
            const ComputeDouble zr2 = zr * zr;
            const ComputeDouble zi2 = zi * zi;
            if (zr2 + zi2 > ESCAPE_RADIUS_SQ)
                break;

            const ComputeDouble zr_temp = zr2 - zi2 + cr;
            zi = zr * zi * 2 + ci;
            zr = zr_temp;
            ++n;
        }
        const int idx = y * width + x;
        image[idx] = n;
    }

    cv::Mat MandelbrotSetCuda::generateImpl() const {
        initialize();

        uchar3 *d_image;
        CHECK_CUDA(hipMalloc(&d_image, width_ * height_ * sizeof(uchar3)));

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width_ + block.x - 1) / block.x, (height_ + block.y - 1) / block.y);

        mandelbrotKernel<<<grid, block>>>(d_image, width_, height_, x_min_, x_max_, y_min_, y_max_);
        CHECK_CUDA(hipGetLastError());

        cv::Mat image(height_, width_, CV_8UC3);
        CHECK_CUDA(hipMemcpy(image.data, d_image, width_ * height_ * sizeof(uchar3), hipMemcpyDeviceToHost));

        CHECK_CUDA(hipFree(d_image));

        return image;
    }

    cv::Mat MandelbrotSetCuda::generateRawMatrix() const {
        // TODO: reuse memory
        float *d_image;
        CHECK_CUDA(hipMalloc(&d_image, width_ * height_ * sizeof(int)));

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width_ + block.x - 1) / block.x, (height_ + block.y - 1) / block.y);

        mandelbrotKernelWithoutColor<<<grid, block>>>(d_image, width_, height_, x_min_, x_max_, y_min_, y_max_);

        cv::Mat image(height_, width_, CV_32FC1);
        CHECK_CUDA(hipMemcpy(image.data, d_image, width_ * height_ * sizeof(int), hipMemcpyDeviceToHost));

        CHECK_CUDA(hipFree(d_image));

        return image;
    }

    cv::Mat MandelbrotSetCuda::detectHighGradient(const cv::Mat &matrix) const {
        cv::Mat normalized;
        cv::normalize(matrix, normalized, 0, 255, cv::NORM_MINMAX, CV_8UC1);

        cv::Mat grad_x, grad_y;
        cv::Sobel(normalized, grad_x, CV_32F, 1, 0);
        cv::Sobel(normalized, grad_y, CV_32F, 0, 1);

        cv::Mat abs_grad_x, abs_grad_y, grad_mag;
        cv::convertScaleAbs(grad_x, abs_grad_x);
        cv::convertScaleAbs(grad_y, abs_grad_y);
        cv::addWeighted(abs_grad_x, 0.5, abs_grad_y, 0.5, 0, grad_mag);

        double min_val, max_val;
        cv::minMaxLoc(grad_mag, &min_val, &max_val);
        double threshold = min_val + (max_val - min_val) * GRADIENT_THRESHOLD;

        cv::Mat mask = grad_mag;
        cv::threshold(grad_mag, mask, threshold, 255, cv::THRESH_BINARY);
        return mask;
    }


} // namespace Mandelbrot
