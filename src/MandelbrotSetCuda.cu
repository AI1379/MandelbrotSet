#include "hip/hip_runtime.h"
//
// Created by Renatus Madrigal on 3/3/2025.
//

#include <opencv2/core.hpp>
#include <opencv2/imgproc.hpp>
#include "ExtendedDouble.cuh"
#include "MandelbrotSetCuda.h"

#define CHECK_CUDA(err)                                                                                                \
    do {                                                                                                               \
        if (err != hipSuccess) {                                                                                      \
            fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));                                              \
            exit(EXIT_FAILURE);                                                                                        \
        }                                                                                                              \
    } while (0)

namespace Mandelbrot {
    constexpr bool USE_EXTENDED_DOUBLE = true;
    using ComputeDouble = std::conditional_t<USE_EXTENDED_DOUBLE, ExtendedDouble, double>;

    __global__ void mandelbrotKernelWithoutColor(float *image, // NOLINT
                                                 size_t width, size_t height, // NOLINT
                                                 double x_min, double x_max, // NOLINT
                                                 double y_min, double y_max) {
        constexpr auto ESCAPE_RADIUS_SQ = MandelbrotSetCuda::ESCAPE_RADIUS_SQ;
        const int x = blockIdx.x * blockDim.x + threadIdx.x;
        const int y = blockIdx.y * blockDim.y + threadIdx.y;
        if (x >= width || y >= height) {
            return;
        }
        const ComputeDouble cr{x_min + (x_max - x_min) * x / width};
        const ComputeDouble ci{y_min + (y_max - y_min) * y / height};

        ComputeDouble zr{0.0}, zi{0.0};
        unsigned int n = 0;
        while (n < MAX_ITERATIONS) {
            const ComputeDouble zr2 = zr * zr;
            const ComputeDouble zi2 = zi * zi;
            if (zr2 + zi2 > ESCAPE_RADIUS_SQ)
                break;

            const ComputeDouble zr_temp = zr2 - zi2 + cr;
            zi = zr * zi * 2 + ci;
            zr = zr_temp;
            ++n;
        }
        const int idx = y * width + x;
        image[idx] = n;
    }

    cv::Mat MandelbrotSetCuda::generateRawMatrixImpl() const {
        // TODO: reuse memory
        float *d_image;
        CHECK_CUDA(hipMalloc(&d_image, width_ * height_ * sizeof(int)));

        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid((width_ + block.x - 1) / block.x, (height_ + block.y - 1) / block.y);

        mandelbrotKernelWithoutColor<<<grid, block>>>(d_image, width_, height_, x_min_, x_max_, y_min_, y_max_);

        cv::Mat image(height_, width_, CV_32FC1);
        CHECK_CUDA(hipMemcpy(image.data, d_image, width_ * height_ * sizeof(int), hipMemcpyDeviceToHost));

        CHECK_CUDA(hipFree(d_image));

        return image;
    }

} // namespace Mandelbrot
