#include "hip/hip_runtime.h"
//
// Created by Renatus Madrigal on 3/4/2025.
//

#include "ExtendedDouble.cuh"

namespace Mandelbrot {


    __host__ __device__ ExtendedDouble::ExtendedDouble(double val) {
        int exp;
        mantissa = frexp(val, &exp);

        // Scale mantissa to [1.0, 2.0)
        if (mantissa != 0.0) {
            mantissa *= 2.0;
            exp--;
        }
        exponent = exp;
    }

    __host__ __device__ void ExtendedDouble::normalize() {
        if (mantissa == 0.0) {
            exponent = 0;
            return;
        }

        int exp_shift;
        mantissa = frexp(mantissa, &exp_shift);
        mantissa *= 2.0;
        exponent += (exp_shift - 1);
    }

    __host__ __device__ ExtendedDouble ExtendedDouble::operator*(const ExtendedDouble &rhs) const {
        ExtendedDouble result;
        result.mantissa = mantissa * rhs.mantissa;
        result.exponent = exponent + rhs.exponent;
        result.normalize();
        return result;
    }

    __host__ __device__ ExtendedDouble ExtendedDouble::operator/(const ExtendedDouble &rhs) const {
        ExtendedDouble result;
        result.mantissa = mantissa / rhs.mantissa;
        result.exponent = exponent - rhs.exponent;
        result.normalize();
        return result;
    }

    __host__ __device__ ExtendedDouble ExtendedDouble::operator+(const ExtendedDouble &rhs) const {
        if (mantissa == 0.0)
            return rhs;
        if (rhs.mantissa == 0.0)
            return *this;

        int max_exp = max(exponent, rhs.exponent);
        double a = mantissa;
        double b = rhs.mantissa;

        if (exponent < max_exp) {
            a = ldexp(a, -(max_exp - exponent));
        } else if (rhs.exponent < max_exp) {
            b = ldexp(b, -(max_exp - rhs.exponent));
        }

        ExtendedDouble result;
        result.mantissa = a + b;
        result.exponent = max_exp;
        result.normalize();
        return result;
    }

    __host__ __device__ ExtendedDouble ExtendedDouble::operator-(const ExtendedDouble &rhs) const {
        if (rhs.mantissa == 0.0)
            return *this;

        int max_exp = max(exponent, rhs.exponent);
        double a = mantissa;
        double b = rhs.mantissa;

        if (exponent < max_exp) {
            a = ldexp(a, -(max_exp - exponent));
        } else if (rhs.exponent < max_exp) {
            b = ldexp(b, -(max_exp - rhs.exponent));
        }

        ExtendedDouble result;
        result.mantissa = a - b;
        result.exponent = max_exp;
        result.normalize();
        return result;
    }

    std::ostream &operator<<(std::ostream &os, const ExtendedDouble &ed) {
        os << static_cast<double>(ed);
        return os;
    }
} // namespace Mandelbrot
