#include "hip/hip_runtime.h"
//
// Created by Renatus Madrigal on 3/4/2025.
//

#include "mandelbrot/ExtendedDouble.cuh"

namespace Mandelbrot {

    __device__ ExtendedDouble operator+(ExtendedDouble a, ExtendedDouble b) {
        if (a.exponent < b.exponent) {
            const ExtendedDouble tmp = a;
            a = b;
            b = tmp;
        }

        int exp_diff = a.exponent - b.exponent;
        if (exp_diff > 53)
            return a;

        double scaled_b_mantissa = __dmul_rd(b.mantissa, __powf(2.0, -exp_diff));
        double new_mantissa = __dadd_rd(a.mantissa, scaled_b_mantissa);

        int shift;
        new_mantissa = frexp(new_mantissa, &shift);
        return {new_mantissa, a.exponent + shift};
    }

} // namespace Mandelbrot
